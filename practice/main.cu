
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

void handleError(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("Error: %s at line %d in file %s\n", hipGetErrorString(error), line, file);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(error) (handleError(error, __FILE__, __LINE__))

int main(void) {
    int c;
    int *c_d;
    HANDLE_ERROR(hipMalloc((void**)&c_d, sizeof(int)));
    HANDLE_ERROR(hipMemcpy(&c, c_d, sizeof(int), hipMemcpyDeviceToHost));
    printf("hello world\n");

    return 0;
}
