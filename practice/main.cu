
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main(void) {
    int c;
    int *c_d;
    hipMalloc((void**)&c_d, sizeof(int));
    hipMemcpy(&c, c_d, sizeof(int), hipMemcpyDeviceToHost);
    printf("hello world\n");
    return 0;
}
