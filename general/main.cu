// #include <time.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int *z, int N) {
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x; 
    if (i < N) {
        z[i] = threadIdx.x;
    }
};

void display(int *z, int N, const unsigned int numBlocks, const unsigned int numThreadsPerBlock) {
    int *z_d;
    hipMalloc((void**)&z_d, sizeof(int)*N);

    // const unsigned int numThreadsPerBlock = 128;
    // const unsigned int numBlocks = (N + numThreadsPerBlock - 1) /numThreadsPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);
    kernel<<<numBlocks, numThreadsPerBlock>>>(z_d, N);
    hipEventRecord(stop, 0);
    hipError_t err = hipGetLastError();  // Check launch errors immediately
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();  // Wait for kernel to finish
    
    hipError_t error = hipEventSynchronize(stop);
    if (error != hipSuccess) {
        printf("ERROR: %s\n", hipGetErrorString(err));
    }

    float timeTaken = 0;
    hipEventElapsedTime(&timeTaken, start, stop);
    // printf("Kernel computation took: %.10f seconds\n", timeTaken/1000);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(z, z_d, sizeof(int)*N, hipMemcpyDeviceToHost);
    hipFree(z_d);
}


int main (int argc, char *argv[]) {

    // int N = (argc > 1) ? atoi(argv[1]) : (1 << 25);
    int N = 128;

    int* z = (int*) malloc(sizeof(int)*N);

    const unsigned int numBlocks = 4;
    const unsigned int numThreadsPerBlock = 32;

    display(z, N, numBlocks, numThreadsPerBlock);


    printf("Total number of threads: %d\n", numBlocks*numThreadsPerBlock);
    printf("Number of blocks: %d\n", numBlocks);
    printf("Number of threads in the blocks: %d\n", numThreadsPerBlock);

    for (int i = 0; i < N; i++) {
        if (i % 16 == 0 && i != 0) {
           printf("\n");
        }
        printf("%3d ", z[i]);
    }
    printf("\n");

    free(z);

    return 0;
}
